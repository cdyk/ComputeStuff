#include "hip/hip_runtime.h"
// This file is part of ComputeStuff copyright (C) 2020 Christopher Dyken.
// Released under the MIT license, please see LICENSE file for details.

#include <hip/hip_runtime_api.h>
#include <glad/gl.h>
#include <GLFW/glfw3.h>
#include <cuda_gl_interop.h>

#include <cmath>
#include <cassert>
#include <vector>
#include <string>
#include <iostream>
#include <fstream>
#include <chrono>

#include <MC.h>

using namespace ComputeStuff::MC;

namespace {

  enum struct FieldFormat : uint32_t
  {
    UInt8,
    UInt16,
    Float
  };

  FieldFormat format = FieldFormat::Float;
  uint32_t nx = 50;
  uint32_t ny = 56;
  uint32_t nz = 71;
  bool wireframe = false;
  bool recreate_context = true;
  bool indexed = true;
  float threshold = 0.f;

  std::vector<char> scalarField_host;

  void onGLFWError(int error, const char* what)
  {
    fprintf(stderr, "GLFW Error: %s\n", what);
  }

  void onKey(GLFWwindow* window, int key, int scancode, int action, int mods)
  {
    bool print_threshold = false;
    if (action == GLFW_PRESS) {
      if (key == GLFW_KEY_W) {
        wireframe = !wireframe;
        fprintf(stderr, "Wireframe: %s\n", wireframe ? "on" : "off");
      }
      else if (key == GLFW_KEY_UP) {
        threshold += 10.f; print_threshold = true;
      }
      else if (key == GLFW_KEY_DOWN) {
        threshold -= 10.f; print_threshold = true;
      }
      else if (key == GLFW_KEY_RIGHT) {
        threshold += 0.01f; print_threshold = true;
      }
      else if (key == GLFW_KEY_LEFT) {
        threshold -= 0.01f; print_threshold = true;
      }
      else if (key == GLFW_KEY_BACKSPACE) {
        threshold = 0.f; print_threshold = true;
      }
      else if (key == GLFW_KEY_I) {
        indexed = !indexed;
        recreate_context = true;
        fprintf(stderr, "Mode is %s", indexed ? "indexed" : "non-indexed");
      }
      if (print_threshold) {
        fprintf(stderr, "Iso-value: %f\n", threshold);
      }
    }
  }


  const std::string simpleVS_src = R"(#version 430
in layout(location=0) vec3 inPosition;
in layout(location=1) vec3 inNormal;
out vec3 normal;
uniform layout(location=0) mat4 MV;
uniform layout(location=1) mat4 MVP;
void main() {
  normal = mat3(MV)*inNormal;
  gl_Position = MVP * vec4(inPosition, 1);
}
)";

  const std::string simpleFS_src = R"(#version 430
in vec3 normal;
out layout(location=0) vec4 outColor;
uniform layout(location=2) vec4 color;
void main() {
  float d = max(0.0, dot(vec3(0,0,1), gl_FrontFacing ? -normal : normal));
  if(gl_FrontFacing)
    outColor = d * color.rgba;
  else
    outColor = color.bgra;
}
)";

  const std::string solidVS_src = R"(#version 430
in layout(location=0) vec3 inPosition;
uniform layout(location=0) mat4 MV;
uniform layout(location=1) mat4 MVP;
void main() {
  gl_Position = MVP * vec4(inPosition, 1);
}
)";

  const std::string solidFS_src = R"(#version 430
out layout(location=0) vec4 outColor;
uniform layout(location=2) vec4 color;
void main() {
  outColor = color.rgba;
}

)";


  [[noreturn]]
  void handleOpenGLError(GLenum error, const std::string file, int line)
  {
    do {
      switch (error) {
      case GL_INVALID_ENUM: fprintf(stderr, "GL_INVALID_ENUM\n"); break;
      case GL_INVALID_VALUE: fprintf(stderr, "GL_INVALID_VALUE\n"); break;
      case GL_INVALID_OPERATION: fprintf(stderr, "GL_INVALID_OPERATION\n"); break;
      case GL_INVALID_FRAMEBUFFER_OPERATION: fprintf(stderr, "GL_INVALID_FRAMEBUFFER_OPERATION\n"); break;
      case GL_OUT_OF_MEMORY: fprintf(stderr, "GL_OUT_OF_MEMORY\n"); break;
      case GL_STACK_OVERFLOW: fprintf(stderr, "GL_STACK_OVERFLOW\n"); break;
      case GL_STACK_UNDERFLOW: fprintf(stderr, "GL_STACK_UNDERFLOW\n"); break;
      default: fprintf(stderr, "Unknown error"); break;
      }
      error = glGetError();
    } while (error != GL_NO_ERROR);
    exit(EXIT_FAILURE);
  }

#define CHECK_GL do { GLenum error = glGetError(); if(error != GL_NO_ERROR) handleOpenGLError(error, __FILE__, __LINE__); } while(0)

  [[noreturn]]
  void handleCudaError(hipError_t error, const std::string file, int line)
  {
    fprintf(stderr, "%s@%d: CUDA: %s\n", file.c_str(), line, hipGetErrorString(error));
    exit(EXIT_FAILURE);
  }

#define CHECK_CUDA do { hipError_t error = hipGetLastError(); if(error != hipSuccess) handleCudaError(error, __FILE__, __LINE__); } while(0)

#define CHECKED_CUDA(a) do { hipError_t error = (a); if(error != hipSuccess) handleCudaError(error, __FILE__, __LINE__); } while(0)

  GLuint createShader(const std::string& src, GLenum shader_type)
  {
    GLuint shader = glCreateShader(shader_type);

    const char* src_array[] = { src.c_str() };
    glShaderSource(shader, 1, src_array, nullptr);
    glCompileShader(shader);

    GLsizei bufSize;
    glGetShaderiv(shader, GL_INFO_LOG_LENGTH, &bufSize);
    if (bufSize) {
      fprintf(stderr, "Source:\n%s", src.c_str());
      std::vector<char> log(bufSize + 1);
      glGetShaderInfoLog(shader, bufSize + 1, nullptr, log.data());
      fprintf(stderr, "Compilator output:\n%s", log.data());
    }

    GLint status;
    glGetShaderiv(shader, GL_COMPILE_STATUS, &status);
    if (status != GL_TRUE) {
      glDeleteShader(shader);
      return 0;
    }
    return shader;
  }

  GLuint createProgram(GLuint VS, GLuint FS)
  {
    GLuint program = glCreateProgram();
    glAttachShader(program, VS);
    glAttachShader(program, FS);
    glLinkProgram(program);


    GLsizei bufSize;
    glGetProgramiv(program, GL_INFO_LOG_LENGTH, &bufSize);
    if (bufSize) {
      std::vector<char> log(bufSize + 1);
      glGetProgramInfoLog(program, bufSize + 1, nullptr, log.data());
      fprintf(stderr, "Linker output:\n%s", log.data());
    }

    GLint status;
    glGetProgramiv(program, GL_LINK_STATUS, &status);
    if (status != GL_TRUE) {
      glDeleteProgram(program);
      return 0;
    }
    return program;
  }

  GLuint createBuffer(GLenum target, GLenum usage, size_t size, const void* data)
  {
    GLuint buffer = 0;
    glGenBuffers(1, &buffer);
    glBindBuffer(target, buffer);
    glBufferData(target, size, data, usage);
    CHECK_GL;
    return buffer;
  }


  void rotMatrixX(float* dst, const float angle)
  {
    const auto c = std::cos(angle);
    const auto s = std::sin(angle);
    dst[4 * 0 + 0] = 1.f; dst[4 * 0 + 1] = 0.f; dst[4 * 0 + 2] = 0.f; dst[4 * 0 + 3] = 0.f;
    dst[4 * 1 + 0] = 0.f; dst[4 * 1 + 1] = c;   dst[4 * 1 + 2] = s;   dst[4 * 1 + 3] = 0.f;
    dst[4 * 2 + 0] = 0.f; dst[4 * 2 + 1] = -s;  dst[4 * 2 + 2] = c;   dst[4 * 2 + 3] = 0.f;
    dst[4 * 3 + 0] = 0.f; dst[4 * 3 + 1] = 0.f; dst[4 * 3 + 2] = 0.f; dst[4 * 3 + 3] = 1.f;
  }

  void rotMatrixY(float* dst, const float angle)
  {
    const auto c = std::cos(angle);
    const auto s = std::sin(angle);
    dst[4 * 0 + 0] = c;   dst[4 * 0 + 1] = 0.f; dst[4 * 0 + 2] = -s;  dst[4 * 0 + 3] = 0.f;
    dst[4 * 1 + 0] = 0.f; dst[4 * 1 + 1] = 1.f; dst[4 * 1 + 2] = 0.f; dst[4 * 1 + 3] = 0.f;
    dst[4 * 2 + 0] = s;   dst[4 * 2 + 1] = 0.f; dst[4 * 2 + 2] = c;   dst[4 * 2 + 3] = 0.f;
    dst[4 * 3 + 0] = 0.f; dst[4 * 3 + 1] = 0.f; dst[4 * 3 + 2] = 0.f; dst[4 * 3 + 3] = 1.f;
  }

  void rotMatrixZ(float* dst, const float angle)
  {
    const auto c = std::cos(angle);
    const auto s = std::sin(angle);
    dst[4 * 0 + 0] = c;   dst[4 * 0 + 1] = s;   dst[4 * 0 + 2] = 0.f; dst[4 * 0 + 3] = 0.f;
    dst[4 * 1 + 0] = -s;  dst[4 * 1 + 1] = c;   dst[4 * 1 + 2] = 0.f; dst[4 * 1 + 3] = 0.f;
    dst[4 * 2 + 0] = 0.f; dst[4 * 2 + 1] = 0.f; dst[4 * 2 + 2] = 1.f; dst[4 * 2 + 3] = 0.f;
    dst[4 * 3 + 0] = 0.f; dst[4 * 3 + 1] = 0.f; dst[4 * 3 + 2] = 0.f; dst[4 * 3 + 3] = 1.f;
  }

  void translateMatrix(float* dst, const float x, const float y, const float z)
  {
    dst[4 * 0 + 0] = 1.f; dst[4 * 0 + 1] = 0.f; dst[4 * 0 + 2] = 0.f; dst[4 * 0 + 3] = 0.f;
    dst[4 * 1 + 0] = 0.f; dst[4 * 1 + 1] = 1.f; dst[4 * 1 + 2] = 0.f; dst[4 * 1 + 3] = 0.f;
    dst[4 * 2 + 0] = 0.f; dst[4 * 2 + 1] = 0.f; dst[4 * 2 + 2] = 1.f; dst[4 * 2 + 3] = 0.f;
    dst[4 * 3 + 0] = x;   dst[4 * 3 + 1] = y;   dst[4 * 3 + 2] = z;   dst[4 * 3 + 3] = 1.f;
  }


  void frustumMatrix(float* dst, const float w, const float h, const float n, const float f)
  {
    auto a = 2.f * n / w;
    auto b = 2.f * n / h;
    auto c = -(f + n) / (f - n);
    auto d = -2.f * f * n / (f - n);
    dst[4 * 0 + 0] = a;   dst[4 * 0 + 1] = 0.f; dst[4 * 0 + 2] = 0.f; dst[4 * 0 + 3] = 0.f;
    dst[4 * 1 + 0] = 0.f; dst[4 * 1 + 1] = b;   dst[4 * 1 + 2] = 0.f; dst[4 * 1 + 3] = 0.f;
    dst[4 * 2 + 0] = 0.f; dst[4 * 2 + 1] = 0.f; dst[4 * 2 + 2] = c;   dst[4 * 2 + 3] = -1.f;
    dst[4 * 3 + 0] = 0.f; dst[4 * 3 + 1] = 0;   dst[4 * 3 + 2] = d;   dst[4 * 3 + 3] = 0.f;
  }


  void matrixMul4(float* D, const float* A, const float* B)
  {
    for (unsigned i = 0; i < 4; i++) {
      for (unsigned j = 0; j < 4; j++) {

        float sum = 0.f;
        for (unsigned k = 0; k < 4; k++) {
          sum += A[4 * k + j] * B[4 * i + k];
        }
        D[4 * i + j] = sum;
      }
    }
  }

  constexpr float cayley(unsigned i, unsigned j, unsigned k, unsigned nx, unsigned ny, unsigned nz)
  {
    float x = (2.f * i) / (nx - 1.f) - 1.f;
    float y = (2.f * j) / (ny - 1.f) - 1.f;
    float z = (2.f * k) / (nz - 1.f) - 1.f;
    float v = 1.f - 16.f * x * y * z - 4.f * (x * x + y * y + z * z);
    return v;
  }

  GLfloat wireBoxVertexData[] =
  {
    0.f, 0.f, 0.f,  1.f, 0.f, 0.f,
    0.f, 0.f, 1.f,  1.f, 0.f, 1.f,
    0.f, 1.f, 0.f,  1.f, 1.f, 0.f,
    0.f, 1.f, 1.f,  1.f, 1.f, 1.f,

    0.f, 0.f, 0.f,  0.f, 1.f, 0.f,
    0.f, 0.f, 1.f,  0.f, 1.f, 1.f,
    1.f, 0.f, 0.f,  1.f, 1.f, 0.f,
    1.f, 0.f, 1.f,  1.f, 1.f, 1.f,

    0.f, 0.f, 0.f,  0.f, 0.f, 1.f,
    0.f, 1.f, 0.f,  0.f, 1.f, 1.f,
    1.f, 0.f, 0.f,  1.f, 0.f, 1.f,
    1.f, 1.f, 0.f,  1.f, 1.f, 1.f
  };

  void buildCayleyField()
  {
    const size_t N = static_cast<size_t>(nx) * ny * nz;
    switch (format) {
    case FieldFormat::UInt8: {
      scalarField_host.resize(N);
      auto* dst = reinterpret_cast<uint8_t*>(scalarField_host.data());
      for (unsigned k = 0; k < nz; k++) {
        for (unsigned j = 0; j < ny; j++) {
          for (unsigned i = 0; i < nx; i++) {
            float v = cayley(i, j, k, nx, ny, nz);
            v = 0.5f * 255.f * (v + 1.f);
            if (v < 0.f) v = 0.f;
            if (255.f < v) v = 255.f;
            *dst++ = static_cast<uint8_t>(v);
          }
        }
      }
      break;
    }
    case FieldFormat::UInt16: {
      scalarField_host.resize(sizeof(uint16_t) * N);
      auto* dst = reinterpret_cast<uint16_t*>(scalarField_host.data());
      for (unsigned k = 0; k < nz; k++) {
        for (unsigned j = 0; j < ny; j++) {
          for (unsigned i = 0; i < nx; i++) {
            float v = cayley(i, j, k, nx, ny, nz);
            v = 0.5f * 65535.f * (v + 1.f);
            if (v < 0.f) v = 0.f;
            if (65535.f < v) v = 65535.f;
            *dst++ = static_cast<uint16_t>(v);
          }
        }
      }
      break;
    }
    case FieldFormat::Float: {
      scalarField_host.resize(sizeof(float) * N);
      auto* dst = reinterpret_cast<float*>(scalarField_host.data());
      for (unsigned k = 0; k < nz; k++) {
        for (unsigned j = 0; j < ny; j++) {
          for (unsigned i = 0; i < nx; i++) {
            *dst++ = cayley(i, j, k, nx, ny, nz);
          }
        }
      }
      break;
    }
    default:
      assert(false && "Unhandled case");
      break;
    }
  }


  bool readFile(const char* path)
  {
    assert(path);
    fprintf(stderr, "Reading %s...\n", path);

    FILE* fp = fopen(path, "rb");
    if (!fp) {
      fprintf(stderr, "Error opening file \"%s\" for reading.\n", path);
      return false;
    }
    if (fseek(fp, 0L, SEEK_END) == 0) {
      uint8_t header[6];
      long size = ftell(fp);
      if (sizeof(header) <= size) {
        if (fseek(fp, 0L, SEEK_SET) == 0) {
          if (fread(header, sizeof(header), 1, fp) == 1) {
            nx = header[0] | header[1] << 8;
            ny = header[2] | header[3] << 8;
            nz = header[4] | header[5] << 8;
            size_t N = static_cast<size_t>(nx) * ny * nz;
            if ((N + 3) * 2 != size) {
              fprintf(stderr, "Unexpected file size.\n");
            }
            else {
              std::vector<uint8_t> tmp(2 * N);
              if (fread(tmp.data(), 2, N, fp) == N) {
                switch (format) {
                case FieldFormat::UInt8: {
                  scalarField_host.resize(N);
                  auto* dst = reinterpret_cast<uint8_t*>(scalarField_host.data());
                  for (size_t i = 0; i < N; i++) {
                    const uint32_t v = tmp[2 * i + 0] | tmp[2 * i + 1] << 8;
                    dst[i] = v >> 4; // 12 bits are in use.
                  }
                  break;
                }
                case FieldFormat::UInt16: {
                  scalarField_host.resize(sizeof(uint16_t) * N);
                  auto* dst = reinterpret_cast<uint16_t*>(scalarField_host.data());
                  for (size_t i = 0; i < N; i++) {
                    const uint32_t v = tmp[2 * i + 0] | tmp[2 * i + 1] << 8;
                    dst[i] = v;
                  }
                  break;
                }
                case FieldFormat::Float: {
                  scalarField_host.resize(sizeof(float) * N);
                  auto* dst = reinterpret_cast<float*>(scalarField_host.data());
                  for (size_t i = 0; i < N; i++) {
                    const uint32_t v = tmp[2 * i + 0] | tmp[2 * i + 1] << 8;
                    dst[i] = static_cast<float>(v);
                  }
                  break;
                }
                default:
                  assert(false && "Unhandled case");
                }
                fprintf(stderr, "Successfully loaded %s\n", path);
                fclose(fp);
                return true;
              }
            }
          }
        }
      }
    }
    fprintf(stderr, "Error loading \"%s\"", path);
    fclose(fp);
    return false;
  }


}



int main(int argc, char** argv)
{
  hipStream_t stream;
  GLFWwindow* win;
  const char* path = nullptr;
  int deviceIndex = 0;

  for (int i = 1; i < argc; i++) {
    if (i + 1 < argc && (strcmp(argv[i], "-d") == 0 || strcmp(argv[i], "--device") == 0)) { deviceIndex = std::atoi(argv[i + 1]); i++; }
    else if (i + 1 < argc && strcmp(argv[i], "-nx") == 0) { nx = uint32_t(std::atoi(argv[i + 1])); i++; }
    else if (i + 1 < argc && strcmp(argv[i], "-ny") == 0) { ny = uint32_t(std::atoi(argv[i + 1])); i++; }
    else if (i + 1 < argc && strcmp(argv[i], "-nz") == 0) { nz = uint32_t(std::atoi(argv[i + 1])); i++; }
    else if (i + 1 < argc && strcmp(argv[i], "-n") == 0) { nx = uint32_t(std::atoi(argv[i + 1])); ny = nx; nz = nx; i++; }
    else if (i + 1 < argc && strcmp(argv[i], "-i") == 0) { threshold = static_cast<float>(std::atof(argv[i + 1])); i++; }
#if 0
    // Currently only float is supported
    else if (i + 1 < argc && strcmp(argv[i], "-f") == 0) {
      if (strcmp(argv[i + 1], "uint8") == 0) { format = FieldFormat::UInt8; }
      else if (strcmp(argv[i + 1], "uint16") == 0) { format = FieldFormat::UInt16; }
      else if (strcmp(argv[i + 1], "float") == 0) { format = FieldFormat::Float; }
      else {
        fprintf(stderr, "Unknown format '%s'", argv[i + 1]);
        return EXIT_FAILURE;
      }
      i++;
    }
#endif
    else if ((strcmp(argv[i], "-h") == 0) || (strcmp(argv[i], "--help") == 0)) {
      fprintf(stderr, "HP5 Marching Cubes test application.\n");
      fprintf(stderr, "Copyright (C) 2020 Christopher Dyken. Released under the MIT license\n\n");
      fprintf(stderr, "Usage: %s [options] [dataset]\n\n", argv[0]);
      fprintf(stderr, "Options:\n");
      fprintf(stderr, "    -d      Choose CUDA device.\n");
      fprintf(stderr, "    -nx     Set number of samples in x direction.\n");
      fprintf(stderr, "    -nx     Set number of samples in y direction.\n");
      fprintf(stderr, "    -nx     Set number of samples in z direction.\n");
      fprintf(stderr, "    -n      Set uniform number of samples in x,y,z directions.\n");
      fprintf(stderr, "    -i      Set iso-value to extract surface for.\n");
      fprintf(stderr, "\nDataset:\n");
      fprintf(stderr, "    cayley    Built-in algebraic surface.\n");
      fprintf(stderr, "    file.dat  Raw binary uint16_t data with three binary uint16_t in front with x,y,z size.\n");
      fprintf(stderr, "\nKey bindings:\n");
      fprintf(stderr, "    right/left  Increase/decrease threshold by 100.\n");
      fprintf(stderr, "    up/down     Increase/decrease threshold by 0.1.\n");
      fprintf(stderr, "    w           Enable/disable wireframe.\n");
      return 0;
    }
    else {
      if (path) {
        fprintf(stderr, "%s: input file already specified\n", argv[i]);
        return EXIT_FAILURE;
      }
      path = argv[i];
    }
  }


  glfwSetErrorCallback(onGLFWError);
  if (!glfwInit()) {
    fprintf(stderr, "GLFW failed to initialize.\n");
    return EXIT_FAILURE;
  }
  glfwWindowHint(GLFW_CONTEXT_VERSION_MAJOR, 4);
  glfwWindowHint(GLFW_CONTEXT_VERSION_MINOR, 3);
  glfwWindowHint(GLFW_OPENGL_PROFILE, GLFW_OPENGL_CORE_PROFILE);

  win = glfwCreateWindow(1280, 720, "Marching cubes test application", nullptr, nullptr);
  glfwSetKeyCallback(win, onKey);
  glfwMakeContextCurrent(win);
  gladLoadGL(glfwGetProcAddress);

  unsigned int deviceCount;
  CHECKED_CUDA(hipGLGetDevices(&deviceCount, nullptr, 0, hipGLDeviceListAll));
  if (deviceCount == 0) {
    fprintf(stderr, "No CUDA-enabled devices available.");
    return EXIT_FAILURE;
  }
  std::vector<int> devices(deviceCount);
  CHECKED_CUDA(hipGLGetDevices(&deviceCount, devices.data(), deviceCount, hipGLDeviceListAll));

  bool found = false;
  for (unsigned k = 0; k < deviceCount; k++) {
    int i = devices[k];
    hipDeviceProp_t dev_prop;
    hipGetDeviceProperties(&dev_prop, i);
    fprintf(stderr, "%c[%i] %s cap=%d.%d\n", i == deviceIndex ? '*' : ' ', i, dev_prop.name, dev_prop.major, dev_prop.minor);
    if (i == deviceIndex) {
      found = true;
    }
  }
  if (!found) {
    fprintf(stderr, "Illegal CUDA device index %d\n", deviceIndex);
    return EXIT_FAILURE;
  }
  hipSetDevice(deviceIndex);
  CHECKED_CUDA(hipStreamCreate(&stream));

  // Set up scalar field
  if (!path) {
    fprintf(stderr, "No input file specified.\n");
    return EXIT_FAILURE;
  }
  else if (strcmp("cayley", path) == 0) {
    buildCayleyField();
  }
  else if (!readFile(path)) {
    return EXIT_FAILURE;
  }
  fprintf(stderr, "Scalar field is [%d x %d x %d] (%d cells total)\n", nx, ny, nz, nx * ny * nz);
  float* deviceMem = nullptr;
  CHECKED_CUDA(hipMalloc(&deviceMem, scalarField_host.size()));
  CHECKED_CUDA(hipMemcpyAsync(deviceMem, scalarField_host.data(), scalarField_host.size(), hipMemcpyHostToDevice, stream));

  assert(nx * ny * nz * 4 == scalarField_host.size());


  auto* tables = createTables(stream);

  GLuint simpleVS = createShader(simpleVS_src, GL_VERTEX_SHADER);
  assert(simpleVS != 0);

  GLuint simpleFS = createShader(simpleFS_src, GL_FRAGMENT_SHADER);
  assert(simpleFS != 0);

  GLuint simplePrg = createProgram(simpleVS, simpleFS);
  assert(simplePrg != 0);

  GLuint solidVS = createShader(solidVS_src, GL_VERTEX_SHADER);
  assert(solidVS != 0);

  GLuint solidFS = createShader(solidFS_src, GL_FRAGMENT_SHADER);
  assert(solidFS != 0);

  GLuint solidPrg = createProgram(solidVS, solidFS);
  assert(solidPrg != 0);


  //GLuint vdatabuf = createBuffer(GL_ARRAY_BUFFER, GL_STATIC_DRAW, sizeof(vertexData), (const void*)vertexData);
  GLuint wireBoxVertexBuffer = createBuffer(GL_ARRAY_BUFFER, GL_STATIC_DRAW, sizeof(wireBoxVertexData),  wireBoxVertexData);
  uint32_t wireBoxVertexCount = sizeof(wireBoxVertexData) / (3 * sizeof(float));

  GLuint wireBoxVbo = 0;
  glGenVertexArrays(1, &wireBoxVbo);
  glBindVertexArray(wireBoxVbo);
  glBindBuffer(GL_ARRAY_BUFFER, wireBoxVertexBuffer);
  glVertexAttribPointer(0, 3, GL_FLOAT, GL_FALSE, sizeof(float) * 3, nullptr);
  glEnableVertexAttribArray(0);

  unsigned eventCounter = 0;
  hipEvent_t events[2 * 4];
  for (size_t i = 0; i < 2 * 4; i++) {
    CHECKED_CUDA(hipEventCreate(&events[i]));
    CHECKED_CUDA(hipEventRecord(events[i], stream));
  }

  GLuint cudaVertexBuf = createBuffer(GL_ARRAY_BUFFER, GL_STREAM_DRAW, 3 * sizeof(float), nullptr);
  hipGraphicsResource* vertexBufferResource = nullptr;
  CHECKED_CUDA(hipGraphicsGLRegisterBuffer(&vertexBufferResource, cudaVertexBuf, hipGraphicsRegisterFlagsWriteDiscard));

  GLuint cudaIndexBuf = createBuffer(GL_ELEMENT_ARRAY_BUFFER, GL_STREAM_DRAW, 3 * sizeof(uint32_t), nullptr);
  hipGraphicsResource* indexBufferResource = nullptr;
  CHECKED_CUDA(hipGraphicsGLRegisterBuffer(&indexBufferResource, cudaIndexBuf, hipGraphicsRegisterFlagsWriteDiscard));

  GLuint cudaVbo = 0;
  glGenVertexArrays(1, &cudaVbo);
  glBindVertexArray(cudaVbo);
  glBindBuffer(GL_ARRAY_BUFFER, cudaVertexBuf);
  glVertexAttribPointer(0, 3, GL_FLOAT, GL_FALSE, sizeof(float) * 6, nullptr);
  glVertexAttribPointer(1, 3, GL_FLOAT, GL_FALSE, sizeof(float) * 6, (void*)(sizeof(float) * 3));
  glEnableVertexAttribArray(0);
  glEnableVertexAttribArray(1);


  auto start = std::chrono::system_clock::now();
  auto timer = std::chrono::high_resolution_clock::now();
  float cuda_ms = 0.f;
  unsigned frames = 0;

  ComputeStuff::MC::Context* ctx = nullptr;
  while (!glfwWindowShouldClose(win)) {
    int width, height;
    glfwGetWindowSize(win, &width, &height);

    uint32_t vertex_count = 0;
    uint32_t index_count = 0;
    {
      if (ctx == nullptr || recreate_context) {
        freeContext(ctx, stream);
        ctx = createContext(tables, make_uint3(nx, ny, nz), indexed, stream);
        recreate_context = false;
      }

      float* cudaVertexBuf_d = nullptr;
      size_t cudaVertexBuf_size = 0;

      uint32_t* cudaIndexBuf_d = nullptr;
      size_t cudaIndexBuf_size = 0;

      CHECKED_CUDA(hipGraphicsMapResources(1, &vertexBufferResource, stream));
      CHECKED_CUDA(hipGraphicsResourceGetMappedPointer((void**)&cudaVertexBuf_d, &cudaVertexBuf_size, vertexBufferResource));
      if (indexed) {
        CHECKED_CUDA(hipGraphicsMapResources(1, &indexBufferResource, stream));
        CHECKED_CUDA(hipGraphicsResourceGetMappedPointer((void**)&cudaIndexBuf_d, &cudaIndexBuf_size, indexBufferResource));
      }

      CHECKED_CUDA(hipEventRecord(events[2 * eventCounter + 0], stream));
      ComputeStuff::MC::buildPN(ctx,
                                cudaVertexBuf_d,
                                cudaIndexBuf_d,
                                cudaVertexBuf_size,
                                cudaIndexBuf_size,
                                nx,
                                nx* ny,
                                make_uint3(0, 0, 0),
                                make_uint3(nx, ny, nz),
                                deviceMem,
                                threshold,
                                stream,
                                true,
                                true);
      CHECKED_CUDA(hipEventRecord(events[2 * eventCounter + 1], stream));
      CHECKED_CUDA(hipGraphicsUnmapResources(1, &vertexBufferResource, stream));
      if (indexed) {
        CHECKED_CUDA(hipGraphicsUnmapResources(1, &indexBufferResource, stream));
      }

      ComputeStuff::MC::getCounts(ctx, &vertex_count, &index_count, stream);
      
      eventCounter = (eventCounter + 1) & 3;
      float ms = 0;
      CHECKED_CUDA(hipEventElapsedTime(&ms, events[2 * eventCounter + 0], events[2 * eventCounter + 1]));
      cuda_ms += ms;

      bool vertexBufTooSmall = cudaVertexBuf_size < 6 * sizeof(float) * vertex_count;
      bool indexBufTooSmall = cudaIndexBuf_size < sizeof(uint32_t)* index_count;

      if (vertexBufTooSmall || indexBufTooSmall) {

        CHECKED_CUDA(hipGraphicsUnregisterResource(vertexBufferResource));
        CHECKED_CUDA(hipGraphicsUnregisterResource(indexBufferResource));

        if (vertexBufTooSmall) {
          size_t newVertexBufSize = 6 * sizeof(float) * (static_cast<size_t>(vertex_count) + vertex_count / 16);
          glBindBuffer(GL_ARRAY_BUFFER, cudaVertexBuf);
          glBufferData(GL_ARRAY_BUFFER, newVertexBufSize, nullptr, GL_STREAM_DRAW);
          glBindBuffer(GL_ARRAY_BUFFER, 0);
          fprintf(stderr, "Resizing: vbuf=%zub\n", newVertexBufSize);
        }

        if (indexBufTooSmall) {
          size_t newIndexBufSize = sizeof(uint32_t) * (index_count + index_count / 16);
          glBindBuffer(GL_ELEMENT_ARRAY_BUFFER, cudaIndexBuf);
          glBufferData(GL_ELEMENT_ARRAY_BUFFER, newIndexBufSize, nullptr, GL_STREAM_DRAW);
          glBindBuffer(GL_ELEMENT_ARRAY_BUFFER, 0);
          fprintf(stderr, "Resizing: ibuf=%zub\n", newIndexBufSize);
        }

        CHECKED_CUDA(hipGraphicsGLRegisterBuffer(&vertexBufferResource, cudaVertexBuf, hipGraphicsRegisterFlagsWriteDiscard));
        CHECKED_CUDA(hipGraphicsGLRegisterBuffer(&indexBufferResource, cudaIndexBuf, hipGraphicsRegisterFlagsWriteDiscard));

        CHECKED_CUDA(hipGraphicsMapResources(1, &vertexBufferResource, stream));
        CHECKED_CUDA(hipGraphicsResourceGetMappedPointer((void**)&cudaVertexBuf_d, &cudaVertexBuf_size, vertexBufferResource));
        if (indexed) {
          CHECKED_CUDA(hipGraphicsMapResources(1, &indexBufferResource, stream));
          CHECKED_CUDA(hipGraphicsResourceGetMappedPointer((void**)&cudaIndexBuf_d, &cudaIndexBuf_size, indexBufferResource));
        }
        fprintf(stderr, "%zu\n", cudaIndexBuf_size);
        ComputeStuff::MC::buildPN(ctx,
                                  cudaVertexBuf_d,
                                  cudaIndexBuf_d,
                                  cudaVertexBuf_size,
                                  cudaIndexBuf_size,
                                  nx,
                                  nx*ny,
                                  make_uint3(0, 0, 0),
                                  make_uint3(nx, ny, nz),
                                  deviceMem,
                                  threshold,
                                  stream,
                                  false,
                                  indexed);
        CHECKED_CUDA(hipGraphicsUnmapResources(1, &vertexBufferResource, stream));
        if (indexed) {
          CHECKED_CUDA(hipGraphicsUnmapResources(1, &indexBufferResource, stream));
        }
      }
    }
    glViewport(0, 0, width, height);

    glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

    std::chrono::duration<double> elapsed = std::chrono::system_clock::now() - start;
    auto seconds = elapsed.count();

    float center[16];
    translateMatrix(center, -0.5f, -0.5f, -0.5f);

    float rx[16];
    rotMatrixX(rx, static_cast<float>(0*1.1 * seconds));

    float ry[16];
    rotMatrixY(ry, static_cast<float>(0*1.7 * seconds));

    float rz[16];
    rotMatrixZ(rz, static_cast<float>(0*1.3 * seconds));

    float shift[16];
    translateMatrix(shift, 0.f, 0.f, -2.0f);

    float frustum[16];
    frustumMatrix(frustum, float(width) / float(height), 1.f, 1.f, 8.f);

    float rx_center[16];
    matrixMul4(rx_center, rx, center);

    float ry_rx[16];
    matrixMul4(ry_rx, ry, rx_center);

    float rz_ry_rx[16];
    matrixMul4(rz_ry_rx, rz, ry_rx);

    float shift_rz_ry_rx[16];
    matrixMul4(shift_rz_ry_rx, shift, rz_ry_rx);

    float frustum_shift_rz_ry_rx[16];
    matrixMul4(frustum_shift_rz_ry_rx, frustum, shift_rz_ry_rx);

    glEnable(GL_DEPTH_TEST);
    glPolygonOffset(0.f, 1.f);
    if (wireframe) {
      glEnable(GL_POLYGON_OFFSET_FILL);
    }
    glBindVertexArray(cudaVbo);
    glPolygonMode(GL_FRONT_AND_BACK, GL_FILL);
    if (indexed) {
      glUseProgram(solidPrg);
      glUniformMatrix4fv(0, 1, GL_FALSE, rz_ry_rx);
      glUniformMatrix4fv(1, 1, GL_FALSE, frustum_shift_rz_ry_rx);
      glUniform4f(2, 0.6f, 0.6f, 0.8f, 1.f);
      glBindBuffer(GL_ELEMENT_ARRAY_BUFFER, cudaIndexBuf);
      glPolygonMode(GL_FRONT_AND_BACK, GL_LINE);
      glDrawElements(GL_TRIANGLES, index_count, GL_UNSIGNED_INT, nullptr);
    }
    else {
      glUseProgram(solidPrg);
      glUniformMatrix4fv(0, 1, GL_FALSE, rz_ry_rx);
      glUniformMatrix4fv(1, 1, GL_FALSE, frustum_shift_rz_ry_rx);
      glUniform4f(2, 0.6f, 0.6f, 0.8f, 1.f);
      glPolygonMode(GL_FRONT_AND_BACK, GL_LINE);
      glDrawArrays(GL_TRIANGLES, 0, vertex_count);
    }
    glDisable(GL_POLYGON_OFFSET_FILL);

#if 0
    if (wireframe) {
      glUseProgram(solidPrg);
      glPolygonMode(GL_FRONT_AND_BACK, GL_LINE);
      glUniformMatrix4fv(0, 1, GL_FALSE, rz_ry_rx);
      glUniformMatrix4fv(1, 1, GL_FALSE, frustum_shift_rz_ry_rx);
      glUniform4f(2, 1.f, 1.f, 1.f, 1.f);
      if (indexed) {
        glDrawElements(GL_LINES, index_count, GL_UNSIGNED_INT, nullptr);
      }
      else {
        glDrawArrays(GL_POINTS, 0, vertex_count);
      }
      glPolygonMode(GL_FRONT_AND_BACK, GL_FILL);
    }
#endif
    glBindBuffer(GL_ELEMENT_ARRAY_BUFFER, 0);

    glBindVertexArray(wireBoxVbo);
    glUseProgram(solidPrg);
    glPolygonMode(GL_FRONT_AND_BACK, GL_LINE);
    glUniformMatrix4fv(0, 1, GL_FALSE, rz_ry_rx);
    glUniformMatrix4fv(1, 1, GL_FALSE, frustum_shift_rz_ry_rx);
    glUniform4f(2, 1.f, 1.f, 1.f, 1.f);
    glDrawArrays(GL_LINES, 0, wireBoxVertexCount);

    glfwSwapBuffers(win);
    glfwPollEvents();

    {
      frames++;
      auto now = std::chrono::high_resolution_clock::now();
      std::chrono::duration<double> elapsed = now - timer;
      auto s = elapsed.count();
      if (10 < frames && 0.5 < s) {
        size_t free, total;
        CHECKED_CUDA(hipMemGetInfo(&free, &total));
        fprintf(stderr, "%.2f FPS (%.2f MVPS) cuda avg: %.2fms (%.2f MVPS) %ux%ux%u Nv=%u Ni=%u ix=%s memfree=%zumb/%zumb\n",
                frames / s, (float(frames)* nx *ny * nz) / (1000000.f * s),
                cuda_ms/frames, (float(frames)* nx* ny* nz) / (1000.f * cuda_ms),
                nx, ny, nz,
                vertex_count,
                index_count,
                indexed ? "y" : "n",
                (free + 1024 * 1024 - 1) / (1024 * 1024),
                (total + 1024 * 1024 - 1) / (1024 * 1024));
        timer = now;
        frames = 0;
        cuda_ms = 0.f;
      }
    }


  }
  glfwDestroyWindow(win);
  glfwTerminate();

  glDeleteShader(simpleVS);

  return EXIT_SUCCESS;
}
