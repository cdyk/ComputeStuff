#include "hip/hip_runtime.h"
// This file is part of ComputeStuff copyright (C) 2020 Christopher Dyken.
// Released under the MIT license, please see LICENSE file for details.

#define USE_NVTOOLS_EXT

#ifdef USE_NVTOOLS_EXT
#include <nvToolsExt.h> 
#endif
#include <hip/hip_runtime_api.h>
#include <glad/gl.h>
#include <GLFW/glfw3.h>
#include <cuda_gl_interop.h>

#include <cmath>
#include <cassert>
#include <vector>
#include <string>
#include <iostream>
#include <fstream>
#include <chrono>

#include <MC.h>

using namespace ComputeStuff::MC;

namespace {

  enum struct FieldFormat : uint32_t
  {
    UInt8,
    UInt16,
    Float
  };

  FieldFormat format = FieldFormat::Float;
  uint3 field_size = make_uint3(256, 256, 256);
  bool wireframe = false;
  bool recreate_context = true;
  bool indexed = true;

  enum LogLevels {
    ALWAYS = 0,
    ERROR = 1,
    WARNING = 2,
    INFO = 3,
    DEBUG = 4,
    TRACE = 5
  };
  uint32_t loglevel = 4;

#define LOG_ALWAYS(msg, ...) do { fputs("[A] ", stderr); fprintf(stderr, msg, ##__VA_ARGS__); fputc('\n', stderr); } while (0)
#define LOG_ERROR(msg, ...) do { if(ERROR <= loglevel) {  fputs("[E] ", stderr); fprintf(stderr, msg, ##__VA_ARGS__); fputc('\n', stderr);} } while (0)
#define LOG_WARNING(msg, ...) do { if(WARNING <= loglevel) {  fputs("[W] ", stderr); fprintf(stderr, msg, ##__VA_ARGS__); fputc('\n', stderr);} } while (0)
#define LOG_INFO(msg, ...) do { if(INFO <= loglevel) {  fputs("[I] ", stderr); fprintf(stderr, msg, ##__VA_ARGS__); fputc('\n', stderr);} } while (0)
#define LOG_DEBUG(msg, ...) do { if(DEBUG <= loglevel) {  fputs("[D] ", stderr); fprintf(stderr, msg, ##__VA_ARGS__); fputc('\n', stderr);} } while (0)
#define LOG_TRACE(msg, ...) do { if(TRACE <= loglevel) {  fputs("[T] ", stderr); fprintf(stderr, msg, ##__VA_ARGS__); fputc('\n', stderr);} } while (0)

  float threshold = 0.f;

  std::vector<char> scalarField_host;

  void onGLFWError(int error, const char* what)
  {
    LOG_ERROR("GLFW Error: %s", what);
  }

  void onKey(GLFWwindow* window, int key, int scancode, int action, int mods)
  {
    bool print_threshold = false;
    if (action == GLFW_PRESS) {
      if (key == GLFW_KEY_W) {
        wireframe = !wireframe;
        LOG_INFO("Wireframe: %s", wireframe ? "on" : "off");
      }
      else if (key == GLFW_KEY_UP) {
        threshold += 10.f; print_threshold = true;
      }
      else if (key == GLFW_KEY_DOWN) {
        threshold -= 10.f; print_threshold = true;
      }
      else if (key == GLFW_KEY_RIGHT) {
        threshold += 0.01f; print_threshold = true;
      }
      else if (key == GLFW_KEY_LEFT) {
        threshold -= 0.01f; print_threshold = true;
      }
      else if (key == GLFW_KEY_BACKSPACE) {
        threshold = 0.f; print_threshold = true;
      }
      else if (key == GLFW_KEY_I) {
        indexed = !indexed;
        recreate_context = true;
        LOG_INFO("Mode is %s", indexed ? "indexed" : "non-indexed");
      }
      if (print_threshold) {
        LOG_INFO("Iso-value: %f", threshold);
      }
    }
  }


  const std::string simpleVS_src = R"(#version 430
in layout(location=0) vec3 inPosition;
in layout(location=1) vec3 inNormal;
out vec3 normal;
uniform layout(location=0) mat4 MV;
uniform layout(location=1) mat4 MVP;
void main() {
  normal = mat3(MV)*inNormal;
  gl_Position = MVP * vec4(inPosition, 1);
}
)";

  const std::string simpleFS_src = R"(#version 430
in vec3 normal;
out layout(location=0) vec4 outColor;
uniform layout(location=2) vec4 color;
void main() {
  float d = max(0.0, dot(vec3(0,0,1), normalize(gl_FrontFacing ? -normal : normal)));
  if(gl_FrontFacing)
    outColor = d * color.rgba;
  else
    outColor = color.bgra;
}
)";

  const std::string solidVS_src = R"(#version 430
in layout(location=0) vec3 inPosition;
uniform layout(location=0) mat4 MV;
uniform layout(location=1) mat4 MVP;
void main() {
  gl_Position = MVP * vec4(inPosition, 1);
}
)";

  const std::string solidFS_src = R"(#version 430
out layout(location=0) vec4 outColor;
uniform layout(location=2) vec4 color;
void main() {
  outColor = color.rgba;
}

)";


  [[noreturn]]
  void handleOpenGLError(GLenum error, const std::string file, int line)
  {
    do {
      switch (error) {
      case GL_INVALID_ENUM: LOG_ERROR("GL_INVALID_ENUM"); break;
      case GL_INVALID_VALUE: LOG_ERROR("GL_INVALID_VALUE"); break;
      case GL_INVALID_OPERATION: LOG_ERROR("GL_INVALID_OPERATION"); break;
      case GL_INVALID_FRAMEBUFFER_OPERATION: LOG_ERROR("GL_INVALID_FRAMEBUFFER_OPERATION"); break;
      case GL_OUT_OF_MEMORY: LOG_ERROR("GL_OUT_OF_MEMORY"); break;
      case GL_STACK_OVERFLOW: LOG_ERROR("GL_STACK_OVERFLOW"); break;
      case GL_STACK_UNDERFLOW: LOG_ERROR("GL_STACK_UNDERFLOW"); break;
      default: LOG_ERROR("Unknown error"); break;
      }
      error = glGetError();
    } while (error != GL_NO_ERROR);
    exit(EXIT_FAILURE);
  }

#define CHECK_GL do { GLenum error = glGetError(); if(error != GL_NO_ERROR) handleOpenGLError(error, __FILE__, __LINE__); } while(0)

  [[noreturn]]
  void handleCudaError(hipError_t error, const std::string file, int line)
  {
    LOG_ERROR("%s@%d: CUDA: %s", file.c_str(), line, hipGetErrorString(error));
    exit(EXIT_FAILURE);
  }

#define CHECK_CUDA do { hipError_t error = hipGetLastError(); if(error != hipSuccess) handleCudaError(error, __FILE__, __LINE__); } while(0)

#define CHECKED_CUDA(a) do { hipError_t error = (a); if(error != hipSuccess) handleCudaError(error, __FILE__, __LINE__); } while(0)

  GLuint createShader(const std::string& src, GLenum shader_type)
  {
    GLuint shader = glCreateShader(shader_type);

    const char* src_array[] = { src.c_str() };
    glShaderSource(shader, 1, src_array, nullptr);
    glCompileShader(shader);

    GLsizei bufSize;
    glGetShaderiv(shader, GL_INFO_LOG_LENGTH, &bufSize);
    if (bufSize) {
      LOG_WARNING("Source:\n%s", src.c_str());
      std::vector<char> log(bufSize + 1);
      glGetShaderInfoLog(shader, bufSize + 1, nullptr, log.data());
      LOG_WARNING("Compilator output:\n%s", log.data());
    }

    GLint status;
    glGetShaderiv(shader, GL_COMPILE_STATUS, &status);
    if (status != GL_TRUE) {
      glDeleteShader(shader);
      return 0;
    }
    return shader;
  }

  GLuint createProgram(GLuint VS, GLuint FS)
  {
    GLuint program = glCreateProgram();
    glAttachShader(program, VS);
    glAttachShader(program, FS);
    glLinkProgram(program);


    GLsizei bufSize;
    glGetProgramiv(program, GL_INFO_LOG_LENGTH, &bufSize);
    if (bufSize) {
      std::vector<char> log(bufSize + 1);
      glGetProgramInfoLog(program, bufSize + 1, nullptr, log.data());
      LOG_WARNING("Linker output:\n%s", log.data());
    }

    GLint status;
    glGetProgramiv(program, GL_LINK_STATUS, &status);
    if (status != GL_TRUE) {
      glDeleteProgram(program);
      return 0;
    }
    return program;
  }

  GLuint createBuffer(GLenum target, GLenum usage, size_t size, const void* data)
  {
    GLuint buffer = 0;
    glGenBuffers(1, &buffer);
    glBindBuffer(target, buffer);
    glBufferData(target, size, data, usage);
    CHECK_GL;
    return buffer;
  }


  void rotMatrixX(float* dst, const float angle)
  {
    const auto c = std::cos(angle);
    const auto s = std::sin(angle);
    dst[4 * 0 + 0] = 1.f; dst[4 * 0 + 1] = 0.f; dst[4 * 0 + 2] = 0.f; dst[4 * 0 + 3] = 0.f;
    dst[4 * 1 + 0] = 0.f; dst[4 * 1 + 1] = c;   dst[4 * 1 + 2] = s;   dst[4 * 1 + 3] = 0.f;
    dst[4 * 2 + 0] = 0.f; dst[4 * 2 + 1] = -s;  dst[4 * 2 + 2] = c;   dst[4 * 2 + 3] = 0.f;
    dst[4 * 3 + 0] = 0.f; dst[4 * 3 + 1] = 0.f; dst[4 * 3 + 2] = 0.f; dst[4 * 3 + 3] = 1.f;
  }

  void rotMatrixY(float* dst, const float angle)
  {
    const auto c = std::cos(angle);
    const auto s = std::sin(angle);
    dst[4 * 0 + 0] = c;   dst[4 * 0 + 1] = 0.f; dst[4 * 0 + 2] = -s;  dst[4 * 0 + 3] = 0.f;
    dst[4 * 1 + 0] = 0.f; dst[4 * 1 + 1] = 1.f; dst[4 * 1 + 2] = 0.f; dst[4 * 1 + 3] = 0.f;
    dst[4 * 2 + 0] = s;   dst[4 * 2 + 1] = 0.f; dst[4 * 2 + 2] = c;   dst[4 * 2 + 3] = 0.f;
    dst[4 * 3 + 0] = 0.f; dst[4 * 3 + 1] = 0.f; dst[4 * 3 + 2] = 0.f; dst[4 * 3 + 3] = 1.f;
  }

  void rotMatrixZ(float* dst, const float angle)
  {
    const auto c = std::cos(angle);
    const auto s = std::sin(angle);
    dst[4 * 0 + 0] = c;   dst[4 * 0 + 1] = s;   dst[4 * 0 + 2] = 0.f; dst[4 * 0 + 3] = 0.f;
    dst[4 * 1 + 0] = -s;  dst[4 * 1 + 1] = c;   dst[4 * 1 + 2] = 0.f; dst[4 * 1 + 3] = 0.f;
    dst[4 * 2 + 0] = 0.f; dst[4 * 2 + 1] = 0.f; dst[4 * 2 + 2] = 1.f; dst[4 * 2 + 3] = 0.f;
    dst[4 * 3 + 0] = 0.f; dst[4 * 3 + 1] = 0.f; dst[4 * 3 + 2] = 0.f; dst[4 * 3 + 3] = 1.f;
  }

  void translateMatrix(float* dst, const float x, const float y, const float z)
  {
    dst[4 * 0 + 0] = 1.f; dst[4 * 0 + 1] = 0.f; dst[4 * 0 + 2] = 0.f; dst[4 * 0 + 3] = 0.f;
    dst[4 * 1 + 0] = 0.f; dst[4 * 1 + 1] = 1.f; dst[4 * 1 + 2] = 0.f; dst[4 * 1 + 3] = 0.f;
    dst[4 * 2 + 0] = 0.f; dst[4 * 2 + 1] = 0.f; dst[4 * 2 + 2] = 1.f; dst[4 * 2 + 3] = 0.f;
    dst[4 * 3 + 0] = x;   dst[4 * 3 + 1] = y;   dst[4 * 3 + 2] = z;   dst[4 * 3 + 3] = 1.f;
  }


  void frustumMatrix(float* dst, const float w, const float h, const float n, const float f)
  {
    auto a = 2.f * n / w;
    auto b = 2.f * n / h;
    auto c = -(f + n) / (f - n);
    auto d = -2.f * f * n / (f - n);
    dst[4 * 0 + 0] = a;   dst[4 * 0 + 1] = 0.f; dst[4 * 0 + 2] = 0.f; dst[4 * 0 + 3] = 0.f;
    dst[4 * 1 + 0] = 0.f; dst[4 * 1 + 1] = b;   dst[4 * 1 + 2] = 0.f; dst[4 * 1 + 3] = 0.f;
    dst[4 * 2 + 0] = 0.f; dst[4 * 2 + 1] = 0.f; dst[4 * 2 + 2] = c;   dst[4 * 2 + 3] = -1.f;
    dst[4 * 3 + 0] = 0.f; dst[4 * 3 + 1] = 0;   dst[4 * 3 + 2] = d;   dst[4 * 3 + 3] = 0.f;
  }


  void matrixMul4(float* D, const float* A, const float* B)
  {
    for (unsigned i = 0; i < 4; i++) {
      for (unsigned j = 0; j < 4; j++) {

        float sum = 0.f;
        for (unsigned k = 0; k < 4; k++) {
          sum += A[4 * k + j] * B[4 * i + k];
        }
        D[4 * i + j] = sum;
      }
    }
  }

  void buildTransforms(float* normal_matrix,
                       float* modelview_projection,
                       const int width,
                       const int height,
                       double seconds)
  {
    float center[16];
    translateMatrix(center, -0.5f, -0.5f, -0.5f);

    float rx[16];
    rotMatrixX(rx, static_cast<float>(0.3 * seconds));

    float ry[16];
    rotMatrixY(ry, static_cast<float>(0.7 * seconds));

    float rz[16];
    rotMatrixZ(rz, static_cast<float>(0.5 * seconds));

    float shift[16];
    translateMatrix(shift, 0.f, 0.f, -2.0f);

    float frustum[16];
    frustumMatrix(frustum, float(width) / float(height), 1.f, 1.f, 8.f);

    float rx_center[16];
    matrixMul4(rx_center, rx, center);

    float ry_rx[16];
    matrixMul4(ry_rx, ry, rx_center);

    matrixMul4(normal_matrix, rz, ry_rx);

    float shift_rz_ry_rx[16];
    matrixMul4(shift_rz_ry_rx, shift, normal_matrix);

    matrixMul4(modelview_projection, frustum, shift_rz_ry_rx);
  }


  constexpr float cayley(unsigned i, unsigned j, unsigned k, uint3 field_size)
  {
    float x = (2.f * i) / (field_size.x - 1.f) - 1.f;
    float y = (2.f * j) / (field_size.y - 1.f) - 1.f;
    float z = (2.f * k) / (field_size.z - 1.f) - 1.f;
    float v = 1.f - 16.f * x * y * z - 4.f * (x * x + y * y + z * z);
    return v;
  }

  GLfloat wireBoxVertexData[] =
  {
    0.f, 0.f, 0.f,  1.f, 0.f, 0.f,
    0.f, 0.f, 1.f,  1.f, 0.f, 1.f,
    0.f, 1.f, 0.f,  1.f, 1.f, 0.f,
    0.f, 1.f, 1.f,  1.f, 1.f, 1.f,

    0.f, 0.f, 0.f,  0.f, 1.f, 0.f,
    0.f, 0.f, 1.f,  0.f, 1.f, 1.f,
    1.f, 0.f, 0.f,  1.f, 1.f, 0.f,
    1.f, 0.f, 1.f,  1.f, 1.f, 1.f,

    0.f, 0.f, 0.f,  0.f, 0.f, 1.f,
    0.f, 1.f, 0.f,  0.f, 1.f, 1.f,
    1.f, 0.f, 0.f,  1.f, 0.f, 1.f,
    1.f, 1.f, 0.f,  1.f, 1.f, 1.f
  };

  void buildCayleyField()
  {
    const size_t N = static_cast<size_t>(field_size.x) * field_size.y * field_size.z;
    switch (format) {
    case FieldFormat::UInt8: {
      scalarField_host.resize(N);
      auto* dst = reinterpret_cast<uint8_t*>(scalarField_host.data());
      for (unsigned k = 0; k < field_size.z; k++) {
        for (unsigned j = 0; j < field_size.y; j++) {
          for (unsigned i = 0; i < field_size.x; i++) {
            float v = cayley(i, j, k, field_size);
            v = 0.5f * 255.f * (v + 1.f);
            if (v < 0.f) v = 0.f;
            if (255.f < v) v = 255.f;
            *dst++ = static_cast<uint8_t>(v);
          }
        }
      }
      break;
    }
    case FieldFormat::UInt16: {
      scalarField_host.resize(sizeof(uint16_t) * N);
      auto* dst = reinterpret_cast<uint16_t*>(scalarField_host.data());
      for (unsigned k = 0; k < field_size.z; k++) {
        for (unsigned j = 0; j < field_size.y; j++) {
          for (unsigned i = 0; i < field_size.x; i++) {
            float v = cayley(i, j, k, field_size);
            v = 0.5f * 65535.f * (v + 1.f);
            if (v < 0.f) v = 0.f;
            if (65535.f < v) v = 65535.f;
            *dst++ = static_cast<uint16_t>(v);
          }
        }
      }
      break;
    }
    case FieldFormat::Float: {
      scalarField_host.resize(sizeof(float) * N);
      auto* dst = reinterpret_cast<float*>(scalarField_host.data());
      for (unsigned k = 0; k < field_size.z; k++) {
        for (unsigned j = 0; j < field_size.y; j++) {
          for (unsigned i = 0; i < field_size.x; i++) {
            *dst++ = cayley(i, j, k, field_size);
          }
        }
      }
      break;
    }
    default:
      assert(false && "Unhandled case");
      break;
    }
  }


  bool readFile(const char* path)
  {
    assert(path);
    LOG_INFO("Reading %s...", path);

    FILE* fp = fopen(path, "rb");
    if (!fp) {
      LOG_ERROR("Error opening file \"%s\" for reading.", path);
      return false;
    }
    if (fseek(fp, 0L, SEEK_END) == 0) {
      uint8_t header[6];
      long size = ftell(fp);
      if (sizeof(header) <= size) {
        if (fseek(fp, 0L, SEEK_SET) == 0) {
          if (fread(header, sizeof(header), 1, fp) == 1) {
            field_size.x = header[0] | header[1] << 8;
            field_size.y = header[2] | header[3] << 8;
            field_size.z = header[4] | header[5] << 8;
            size_t N = static_cast<size_t>(field_size.x) * field_size.y * field_size.z;
            if ((N + 3) * 2 != size) {
              LOG_ERROR("Unexpected file size.");
            }
            else {
              std::vector<uint8_t> tmp(2 * N);
              if (fread(tmp.data(), 2, N, fp) == N) {
                switch (format) {
                case FieldFormat::UInt8: {
                  scalarField_host.resize(N);
                  auto* dst = reinterpret_cast<uint8_t*>(scalarField_host.data());
                  for (size_t i = 0; i < N; i++) {
                    const uint32_t v = tmp[2 * i + 0] | tmp[2 * i + 1] << 8;
                    dst[i] = v >> 4; // 12 bits are in use.
                  }
                  break;
                }
                case FieldFormat::UInt16: {
                  scalarField_host.resize(sizeof(uint16_t) * N);
                  auto* dst = reinterpret_cast<uint16_t*>(scalarField_host.data());
                  for (size_t i = 0; i < N; i++) {
                    const uint32_t v = tmp[2 * i + 0] | tmp[2 * i + 1] << 8;
                    dst[i] = v;
                  }
                  break;
                }
                case FieldFormat::Float: {
                  scalarField_host.resize(sizeof(float) * N);
                  auto* dst = reinterpret_cast<float*>(scalarField_host.data());
                  for (size_t i = 0; i < N; i++) {
                    const uint32_t v = tmp[2 * i + 0] | tmp[2 * i + 1] << 8;
                    dst[i] = static_cast<float>(v);
                  }
                  break;
                }
                default:
                  assert(false && "Unhandled case");
                }
                LOG_INFO("Successfully loaded %s", path);
                fclose(fp);
                return true;
              }
            }
          }
        }
      }
    }
    LOG_ERROR("Error loading \"%s\"", path);
    fclose(fp);
    return false;
  }

  void setupScalarField(float*& scalar_field_d, const char* path, const uint3& field_size, hipStream_t stream)
  {
    // Set up scalar field
    if (!path) {
      LOG_ERROR("No input file specified.");
      exit(EXIT_FAILURE);
    }
    else if (strcmp("cayley", path) == 0) {
      buildCayleyField();
    }
    else if (!readFile(path)) {
      exit(EXIT_FAILURE);
    }
    assert(static_cast<size_t>(field_size.x) * field_size.y * field_size.z * 4 == scalarField_host.size());
    LOG_INFO("Scalar field is [%d x %d x %d] (%d cells total)", field_size.x, field_size.y, field_size.z, field_size.x * field_size.y * field_size.z);
    CHECKED_CUDA(hipMalloc(&scalar_field_d, scalarField_host.size()));
    CHECKED_CUDA(hipMemcpyAsync(scalar_field_d, scalarField_host.data(), scalarField_host.size(), hipMemcpyHostToDevice, stream));
  }

  void initWindowAndGL(GLFWwindow*& win, GLuint& shadedProg, GLuint& solidProg)
  {
    glfwSetErrorCallback(onGLFWError);
    if (!glfwInit()) {
      LOG_ERROR("GLFW failed to initialize.");
      exit(EXIT_FAILURE);
    }
    glfwWindowHint(GLFW_CONTEXT_VERSION_MAJOR, 4);
    glfwWindowHint(GLFW_CONTEXT_VERSION_MINOR, 3);
    glfwWindowHint(GLFW_OPENGL_PROFILE, GLFW_OPENGL_CORE_PROFILE);

    win = glfwCreateWindow(1280, 720, "Marching cubes test application", nullptr, nullptr);
    glfwSetKeyCallback(win, onKey);
    glfwMakeContextCurrent(win);
    gladLoadGL(glfwGetProcAddress);

    GLuint simpleVS = createShader(simpleVS_src, GL_VERTEX_SHADER);   assert(simpleVS != 0);
    GLuint simpleFS = createShader(simpleFS_src, GL_FRAGMENT_SHADER); assert(simpleFS != 0);
    shadedProg = createProgram(simpleVS, simpleFS);                   assert(shadedProg != 0);

    GLuint solidVS = createShader(solidVS_src, GL_VERTEX_SHADER);     assert(solidVS != 0);
    GLuint solidFS = createShader(solidFS_src, GL_FRAGMENT_SHADER);   assert(solidFS != 0);
    solidProg = createProgram(solidVS, solidFS);                      assert(solidProg != 0);
  }

}



int main(int argc, char** argv)
{
  hipStream_t stream;
  const char* path = nullptr;
  int deviceIndex = 0;
  bool benchmark = false;

  for (int i = 1; i < argc; i++) {
    if (i + 1 < argc && (strcmp(argv[i], "-d") == 0 || strcmp(argv[i], "--device") == 0)) { deviceIndex = std::atoi(argv[i + 1]); i++; }
    else if (i + 1 < argc && strcmp(argv[i], "-nx") == 0) { field_size.x = uint32_t(std::atoi(argv[i + 1])); i++; }
    else if (i + 1 < argc && strcmp(argv[i], "-ny") == 0) { field_size.y = uint32_t(std::atoi(argv[i + 1])); i++; }
    else if (i + 1 < argc && strcmp(argv[i], "-nz") == 0) { field_size.z = uint32_t(std::atoi(argv[i + 1])); i++; }
    else if (i + 1 < argc && strcmp(argv[i], "-n") == 0) { field_size.x = uint32_t(std::atoi(argv[i + 1])); field_size.y = field_size.x; field_size.z = field_size.x; i++; }
    else if (i + 1 < argc && strcmp(argv[i], "-i") == 0) { threshold = static_cast<float>(std::atof(argv[i + 1])); i++; }
    else if (i + 1 < argc && strcmp(argv[i], "-l") == 0) { loglevel = uint32_t(std::atoi(argv[i + 1])); i++; }
#if 0
    // Currently only float is supported
    else if (i + 1 < argc && strcmp(argv[i], "-f") == 0) {
      if (strcmp(argv[i + 1], "uint8") == 0) { format = FieldFormat::UInt8; }
      else if (strcmp(argv[i + 1], "uint16") == 0) { format = FieldFormat::UInt16; }
      else if (strcmp(argv[i + 1], "float") == 0) { format = FieldFormat::Float; }
      else {
        fprintf(stderr, "Unknown format '%s'", argv[i + 1]);
        return EXIT_FAILURE;
      }
      i++;
    }
#endif
    else if ((strcmp(argv[i], "-b") == 0) || (strcmp(argv[i], "--benchmark") == 0)) { benchmark = true; }
    else if ((strcmp(argv[i], "-h") == 0) || (strcmp(argv[i], "--help") == 0)) {
      fprintf(stderr, "HP5 Marching Cubes test application.\n");
      fprintf(stderr, "Copyright (C) 2020 Christopher Dyken. Released under the MIT license\n\n");
      fprintf(stderr, "Usage: %s [options] [dataset]\n\n", argv[0]);
      fprintf(stderr, "Options:\n");
      fprintf(stderr, "    -d   int    Choose CUDA device.\n");
      fprintf(stderr, "    -nx  int    Set number of samples in x direction.\n");
      fprintf(stderr, "    -nx  int    Set number of samples in y direction.\n");
      fprintf(stderr, "    -nx  int    Set number of samples in z direction.\n");
      fprintf(stderr, "    -n   int    Set uniform number of samples in x,y,z directions.\n");
      fprintf(stderr, "    -i   float  Set iso-value to extract surface for.\n");
      fprintf(stderr, "    -l   int    Log-level, higher is more verbose.\n");
      fprintf(stderr, "    -b          Enable benchmark mode without OpenGL interop.\n");
      fprintf(stderr, "\nDataset:\n");
      fprintf(stderr, "    cayley    Built-in algebraic surface.\n");
      fprintf(stderr, "    file.dat  Raw binary uint16_t data with three binary uint16_t in front with x,y,z size.\n");
      fprintf(stderr, "\nKey bindings:\n");
      fprintf(stderr, "    right/left  Increase/decrease threshold by 100.\n");
      fprintf(stderr, "    up/down     Increase/decrease threshold by 0.1.\n");
      fprintf(stderr, "    w           Enable/disable wireframe.\n");
      return 0;
    }
    else {
      if (path) {
        LOG_ERROR("%s: input already specified", argv[i]);
        return EXIT_FAILURE;
      }
      path = argv[i];
    }
  }

  if (benchmark) {
    
    int deviceCount = 0;
    CHECKED_CUDA(hipGetDeviceCount(&deviceCount));

    bool found = false;
    for (int i = 0; i < deviceCount; i++) {
      hipDeviceProp_t dev_prop;
      hipGetDeviceProperties(&dev_prop, i);
      LOG_INFO("%c[%i] %s cap=%d.%d", i == deviceIndex ? '*' : ' ', i, dev_prop.name, dev_prop.major, dev_prop.minor);
      if (i == deviceIndex) {
        found = true;
      }
    }
    if (!found) {
      LOG_ERROR("Illegal CUDA device index %d", deviceIndex);
      return EXIT_FAILURE;
    }
    hipSetDevice(deviceIndex);
    CHECKED_CUDA(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));

    // Create events for timing
    static const unsigned eventNum = 32;
    hipEvent_t events[2 * eventNum];
    for (size_t i = 0; i < 2 * eventNum; i++) {
      CHECKED_CUDA(hipEventCreate(&events[i]));
      CHECKED_CUDA(hipEventRecord(events[i], stream));
    }

    size_t free, total;
    CHECKED_CUDA(hipMemGetInfo(&free, &total));
    LOG_INFO("CUDA memory free=%zumb total=%zumb", (free + 1024 * 1024 - 1) / (1024 * 1024), (total + 1024 * 1024 - 1) / (1024 * 1024));

    float* scalar_field_d = nullptr;
    setupScalarField(scalar_field_d, path, field_size, stream);
    LOG_INFO("Built scalar field");

    CHECKED_CUDA(hipMemGetInfo(&free, &total));
    LOG_INFO("CUDA memory free=%zumb total=%zumb", (free + 1024 * 1024 - 1) / (1024 * 1024), (total + 1024 * 1024 - 1) / (1024 * 1024));

    auto* tables = createTables(stream);

    struct {
      const char* name;
      bool indexed;
      bool sync;
    }
    benchmark_cases[] = {
      {"ix sync", true, true},
      {"noix sync", false, true},
      {"ix nosync", true, false},
      {"noix nosync", false, false}
    };

    float min_time = 0.5;
    for (auto& bc : benchmark_cases) {
#ifdef USE_NVTOOLS_EXT
      nvtxRangePush(bc.name);
#endif
      auto* ctx = createContext(tables, field_size, true, stream);
      LOG_INFO("%12s: Created context.", bc.name);

      // Run with no output buffers to get size of output.
      ComputeStuff::MC::buildPN(ctx,
                                nullptr,
                                nullptr,
                                0,
                                0,
                                field_size.x,
                                field_size.x* field_size.y,
                                make_uint3(0, 0, 0),
                                field_size,
                                scalar_field_d,
                                threshold,
                                stream,
                                true,
                                true);
      uint32_t vertex_count = 0;
      uint32_t index_count = 0;
      ComputeStuff::MC::getCounts(ctx, &vertex_count, &index_count, stream);

      float* vertex_data_d = nullptr;
      CHECKED_CUDA(hipMalloc(&vertex_data_d, 6 * sizeof(float) * vertex_count));
      uint32_t* index_data_d = nullptr;
      CHECKED_CUDA(hipMalloc(&index_data_d, sizeof(uint32_t)* index_count));
      LOG_INFO("%12s: Allocated output buffers.", bc.name);

      LOG_INFO("%12s: Warming up", bc.name);
      for (unsigned i = 0; i < 100; i++) {
        ComputeStuff::MC::buildPN(ctx,
                                  vertex_data_d,
                                  index_data_d,
                                  6 * sizeof(float) * vertex_count,
                                  sizeof(uint32_t) * index_count,
                                  field_size.x,
                                  field_size.x * field_size.y,
                                  make_uint3(0, 0, 0),
                                  field_size,
                                  scalar_field_d,
                                  threshold,
                                  stream,
                                  true,
                                  true);
        if (bc.sync) {
          ComputeStuff::MC::getCounts(ctx, &vertex_count, &index_count, stream);
        }
      }

      LOG_INFO("%12s: Benchmarking", bc.name);
      auto start = std::chrono::high_resolution_clock::now();
      double elapsed = 0.f;
      float cuda_ms = 0.f;
      unsigned iterations = 0;
      unsigned cuda_ms_n = 0;
#ifdef USE_NVTOOLS_EXT
      nvtxRangePush("Benchmark runs");
#endif
      while (iterations < 100 || elapsed < min_time) {
        CHECKED_CUDA(hipEventRecord(events[2 * (iterations % eventNum) + 0], stream));
        ComputeStuff::MC::buildPN(ctx,
                                  vertex_data_d,
                                  index_data_d,
                                  6 * sizeof(float) * vertex_count,
                                  sizeof(uint32_t) * index_count,
                                  field_size.x,
                                  field_size.x * field_size.y,
                                  make_uint3(0, 0, 0),
                                  field_size,
                                  scalar_field_d,
                                  threshold,
                                  stream,
                                  true,
                                  true);
        if (bc.sync) {
          ComputeStuff::MC::getCounts(ctx, &vertex_count, &index_count, stream);
        }
        CHECKED_CUDA(hipEventRecord(events[2 * (iterations % eventNum) + 1], stream));

        if (eventNum <= iterations) {
          float ms = 0;
          if (!bc.sync) {
            CHECKED_CUDA(hipEventSynchronize(events[2 * ((iterations + 1) % eventNum) + 1]));
          }

          CHECKED_CUDA(hipEventElapsedTime(&ms,
                                            events[2 * ((iterations + 1) % eventNum) + 0],
                                            events[2 * ((iterations + 1) % eventNum) + 1]));
          cuda_ms += ms;
          cuda_ms_n++;
        }

        std::chrono::duration<double> span = std::chrono::high_resolution_clock::now() - start;
        elapsed = span.count();
        iterations++;
      }
#ifdef USE_NVTOOLS_EXT
      nvtxRangePop();
#endif
      CHECKED_CUDA(hipMemGetInfo(&free, &total));
      LOG_ALWAYS("%12s: %.2f FPS (%.0fMVPS) cuda: %.2fms (%.0f MVPS) %ux%ux%u Nv=%u Ni=%u memfree=%zumb/%zumb",
              bc.name,
              iterations / elapsed, (float(iterations) * field_size.x * field_size.y * field_size.z) / (1000000.f * elapsed),
              cuda_ms / cuda_ms_n, (float(cuda_ms_n) * field_size.x * field_size.y * field_size.z) / (1000.f * cuda_ms),
              field_size.x, field_size.y, field_size.z,
              vertex_count,
              index_count,
              (free + 1024 * 1024 - 1) / (1024 * 1024),
              (total + 1024 * 1024 - 1) / (1024 * 1024));

      freeContext(ctx, stream);
      CHECKED_CUDA(hipStreamSynchronize(stream));
      CHECKED_CUDA(hipFree(vertex_data_d));
      CHECKED_CUDA(hipFree(index_data_d));

      CHECKED_CUDA(hipMemGetInfo(&free, &total));
      LOG_INFO("%12s: Released resources free=%zumb total=%zumb", bc.name, (free + 1024 * 1024 - 1) / (1024 * 1024), (total + 1024 * 1024 - 1) / (1024 * 1024));
#ifdef USE_NVTOOLS_EXT
      nvtxRangePop();
#endif
    }

    LOG_ALWAYS("Exiting...");
    CHECKED_CUDA(hipMemGetInfo(&free, &total));
    LOG_INFO("CUDA memory free=%zumb total=%zumb", (free + 1024 * 1024 - 1) / (1024 * 1024), (total + 1024 * 1024 - 1) / (1024 * 1024));
    return 0;
  }

  GLFWwindow* win = nullptr;
  GLuint shadedProg = 0;
  GLuint solidProg = 0;
  initWindowAndGL(win, shadedProg, solidProg);

  unsigned int deviceCount;
  CHECKED_CUDA(hipGLGetDevices(&deviceCount, nullptr, 0, hipGLDeviceListAll));
  if (deviceCount == 0) {
    LOG_ERROR("No CUDA-enabled devices available.");
    return EXIT_FAILURE;
  }
  std::vector<int> devices(deviceCount);
  CHECKED_CUDA(hipGLGetDevices(&deviceCount, devices.data(), deviceCount, hipGLDeviceListAll));

  bool found = false;
  for (unsigned k = 0; k < deviceCount; k++) {
    int i = devices[k];
    hipDeviceProp_t dev_prop;
    hipGetDeviceProperties(&dev_prop, i);
    LOG_INFO("%c[%i] %s cap=%d.%d", i == deviceIndex ? '*' : ' ', i, dev_prop.name, dev_prop.major, dev_prop.minor);
    if (i == deviceIndex) {
      found = true;
    }
  }
  if (!found) {
    LOG_ERROR("Illegal CUDA device index %d", deviceIndex);
    return EXIT_FAILURE;
  }
  hipSetDevice(deviceIndex);
  CHECKED_CUDA(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));

  // Set up scalar field
  float* scalar_field_d = nullptr;
  setupScalarField(scalar_field_d, path, field_size, stream);

  auto* tables = createTables(stream);

  GLuint wireBoxVertexBuffer = createBuffer(GL_ARRAY_BUFFER, GL_STATIC_DRAW, sizeof(wireBoxVertexData),  wireBoxVertexData);
  uint32_t wireBoxVertexCount = sizeof(wireBoxVertexData) / (3 * sizeof(float));
  GLuint wireBoxVbo = 0;
  glGenVertexArrays(1, &wireBoxVbo);
  glBindVertexArray(wireBoxVbo);
  glBindBuffer(GL_ARRAY_BUFFER, wireBoxVertexBuffer);
  glVertexAttribPointer(0, 3, GL_FLOAT, GL_FALSE, sizeof(float) * 3, nullptr);
  glEnableVertexAttribArray(0);

  unsigned eventCounter = 0;
  hipEvent_t events[2 * 4];
  for (size_t i = 0; i < 2 * 4; i++) {
    CHECKED_CUDA(hipEventCreate(&events[i]));
    CHECKED_CUDA(hipEventRecord(events[i], stream));
  }

  GLuint cudaVertexBuf = createBuffer(GL_ARRAY_BUFFER, GL_STREAM_DRAW, 3 * sizeof(float), nullptr);
  hipGraphicsResource* vertexBufferResource = nullptr;
  CHECKED_CUDA(hipGraphicsGLRegisterBuffer(&vertexBufferResource, cudaVertexBuf, hipGraphicsRegisterFlagsWriteDiscard));

  GLuint cudaIndexBuf = createBuffer(GL_ELEMENT_ARRAY_BUFFER, GL_STREAM_DRAW, 3 * sizeof(uint32_t), nullptr);
  hipGraphicsResource* indexBufferResource = nullptr;
  CHECKED_CUDA(hipGraphicsGLRegisterBuffer(&indexBufferResource, cudaIndexBuf, hipGraphicsRegisterFlagsWriteDiscard));

  GLuint cudaVbo = 0;
  glGenVertexArrays(1, &cudaVbo);
  glBindVertexArray(cudaVbo);
  glBindBuffer(GL_ARRAY_BUFFER, cudaVertexBuf);
  glVertexAttribPointer(0, 3, GL_FLOAT, GL_FALSE, sizeof(float) * 6, nullptr);
  glVertexAttribPointer(1, 3, GL_FLOAT, GL_FALSE, sizeof(float) * 6, (void*)(sizeof(float) * 3));
  glEnableVertexAttribArray(0);
  glEnableVertexAttribArray(1);


  auto start = std::chrono::system_clock::now();
  auto timer = std::chrono::high_resolution_clock::now();
  float cuda_ms = 0.f;
  unsigned frames = 0u;

  ComputeStuff::MC::Context* ctx = nullptr;
  while (!glfwWindowShouldClose(win)) {
    int width, height;
    glfwGetWindowSize(win, &width, &height);

    uint32_t vertex_count = 0;
    uint32_t index_count = 0;
    {
      if (ctx == nullptr || recreate_context) {
        freeContext(ctx, stream);
        ctx = createContext(tables, field_size, indexed, stream);
        recreate_context = false;
      }

      float* cudaVertexBuf_d = nullptr;
      size_t cudaVertexBuf_size = 0;

      uint32_t* cudaIndexBuf_d = nullptr;
      size_t cudaIndexBuf_size = 0;

      CHECKED_CUDA(hipGraphicsMapResources(1, &vertexBufferResource, stream));
      CHECKED_CUDA(hipGraphicsResourceGetMappedPointer((void**)&cudaVertexBuf_d, &cudaVertexBuf_size, vertexBufferResource));
      if (indexed) {
        CHECKED_CUDA(hipGraphicsMapResources(1, &indexBufferResource, stream));
        CHECKED_CUDA(hipGraphicsResourceGetMappedPointer((void**)&cudaIndexBuf_d, &cudaIndexBuf_size, indexBufferResource));
      }
      CHECKED_CUDA(hipEventRecord(events[2 * eventCounter + 0], stream));
      ComputeStuff::MC::buildPN(ctx,
                                cudaVertexBuf_d,
                                cudaIndexBuf_d,
                                cudaVertexBuf_size,
                                cudaIndexBuf_size,
                                field_size.x,
                                field_size.x* field_size.y,
                                make_uint3(0, 0, 0),
                                field_size,
                                scalar_field_d,
                                threshold,
                                stream,
                                true,
                                true);
      CHECKED_CUDA(hipEventRecord(events[2 * eventCounter + 1], stream));
      CHECKED_CUDA(hipGraphicsUnmapResources(1, &vertexBufferResource, stream));
      if (indexed) {
        CHECKED_CUDA(hipGraphicsUnmapResources(1, &indexBufferResource, stream));
      }

      ComputeStuff::MC::getCounts(ctx, &vertex_count, &index_count, stream);
      
      eventCounter = (eventCounter + 1) & 3;
      float ms = 0;
      CHECKED_CUDA(hipEventElapsedTime(&ms, events[2 * eventCounter + 0], events[2 * eventCounter + 1]));
      cuda_ms += ms;

      bool vertexBufTooSmall = cudaVertexBuf_size < 6 * sizeof(float) * vertex_count;
      bool indexBufTooSmall = cudaIndexBuf_size < sizeof(uint32_t)* index_count;

      if (vertexBufTooSmall || indexBufTooSmall) {

        CHECKED_CUDA(hipGraphicsUnregisterResource(vertexBufferResource));
        CHECKED_CUDA(hipGraphicsUnregisterResource(indexBufferResource));

        if (vertexBufTooSmall) {
          size_t newVertexBufSize = 6 * sizeof(float) * (static_cast<size_t>(vertex_count) + vertex_count / 16);
          glBindBuffer(GL_ARRAY_BUFFER, cudaVertexBuf);
          glBufferData(GL_ARRAY_BUFFER, newVertexBufSize, nullptr, GL_STREAM_DRAW);
          glBindBuffer(GL_ARRAY_BUFFER, 0);
          LOG_INFO("Resizing: vbuf=%zub", newVertexBufSize);
        }

        if (indexBufTooSmall) {
          size_t newIndexBufSize = sizeof(uint32_t) * (index_count + index_count / 16);
          glBindBuffer(GL_ELEMENT_ARRAY_BUFFER, cudaIndexBuf);
          glBufferData(GL_ELEMENT_ARRAY_BUFFER, newIndexBufSize, nullptr, GL_STREAM_DRAW);
          glBindBuffer(GL_ELEMENT_ARRAY_BUFFER, 0);
          LOG_INFO("Resizing: ibuf=%zub", newIndexBufSize);
        }

        CHECKED_CUDA(hipGraphicsGLRegisterBuffer(&vertexBufferResource, cudaVertexBuf, hipGraphicsRegisterFlagsWriteDiscard));
        CHECKED_CUDA(hipGraphicsGLRegisterBuffer(&indexBufferResource, cudaIndexBuf, hipGraphicsRegisterFlagsWriteDiscard));

        CHECKED_CUDA(hipGraphicsMapResources(1, &vertexBufferResource, stream));
        CHECKED_CUDA(hipGraphicsResourceGetMappedPointer((void**)&cudaVertexBuf_d, &cudaVertexBuf_size, vertexBufferResource));
        if (indexed) {
          CHECKED_CUDA(hipGraphicsMapResources(1, &indexBufferResource, stream));
          CHECKED_CUDA(hipGraphicsResourceGetMappedPointer((void**)&cudaIndexBuf_d, &cudaIndexBuf_size, indexBufferResource));
        }
        ComputeStuff::MC::buildPN(ctx,
                                  cudaVertexBuf_d,
                                  cudaIndexBuf_d,
                                  cudaVertexBuf_size,
                                  cudaIndexBuf_size,
                                  field_size.x,
                                  field_size.x* field_size.y,
                                  make_uint3(0, 0, 0),
                                  field_size,
                                  scalar_field_d,
                                  threshold,
                                  stream,
                                  false,
                                  indexed);
        CHECKED_CUDA(hipGraphicsUnmapResources(1, &vertexBufferResource, stream));
        if (indexed) {
          CHECKED_CUDA(hipGraphicsUnmapResources(1, &indexBufferResource, stream));
        }
      }
    }
    glViewport(0, 0, width, height);

    glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

    std::chrono::duration<double> elapsed = std::chrono::system_clock::now() - start;

    float normal_matrix[16];
    float modelview_projection[16];
    buildTransforms(normal_matrix, modelview_projection, width, height, elapsed.count());


    glEnable(GL_DEPTH_TEST);
    glPolygonOffset(0.f, 1.f);
    if (wireframe) {
      glEnable(GL_POLYGON_OFFSET_FILL);
    }
    glBindVertexArray(cudaVbo);
    glPolygonMode(GL_FRONT_AND_BACK, GL_FILL);
    glUseProgram(shadedProg);
    glUniformMatrix4fv(0, 1, GL_FALSE, normal_matrix);
    glUniformMatrix4fv(1, 1, GL_FALSE, modelview_projection);
    glUniform4f(2, 0.6f, 0.6f, 0.8f, 1.f);
    if (indexed) {
      glBindBuffer(GL_ELEMENT_ARRAY_BUFFER, cudaIndexBuf);
      glDrawElements(GL_TRIANGLES, index_count, GL_UNSIGNED_INT, nullptr);
    }
    else {
      glDrawArrays(GL_TRIANGLES, 0, vertex_count);
    }
    glDisable(GL_POLYGON_OFFSET_FILL);


    if (wireframe) {
      glUseProgram(solidProg);
      glPolygonMode(GL_FRONT_AND_BACK, GL_LINE);
      glUniformMatrix4fv(0, 1, GL_FALSE, normal_matrix);
      glUniformMatrix4fv(1, 1, GL_FALSE, modelview_projection);
      glUniform4f(2, 1.f, 1.f, 1.f, 1.f);
      if (indexed) {
        glBindBuffer(GL_ELEMENT_ARRAY_BUFFER, cudaIndexBuf);
        glDrawElements(GL_TRIANGLES, index_count, GL_UNSIGNED_INT, nullptr);
      }
      else {
        glDrawArrays(GL_TRIANGLES, 0, vertex_count);
      }
      glPolygonMode(GL_FRONT_AND_BACK, GL_FILL);
    }

    glBindBuffer(GL_ELEMENT_ARRAY_BUFFER, 0);

    glBindVertexArray(wireBoxVbo);
    glUseProgram(solidProg);
    glPolygonMode(GL_FRONT_AND_BACK, GL_LINE);
    glUniformMatrix4fv(0, 1, GL_FALSE, normal_matrix);
    glUniformMatrix4fv(1, 1, GL_FALSE, modelview_projection);
    glUniform4f(2, 1.f, 1.f, 1.f, 1.f);
    glDrawArrays(GL_LINES, 0, wireBoxVertexCount);

    glfwSwapBuffers(win);
    glfwPollEvents();

    {
      frames++;
      auto now = std::chrono::high_resolution_clock::now();
      std::chrono::duration<double> elapsed = now - timer;
      auto s = elapsed.count();
      if (10 < frames && 3.0 < s) {
        size_t free, total;
        CHECKED_CUDA(hipMemGetInfo(&free, &total));
        LOG_INFO("%.2f FPS (%.2f MVPS) cuda avg: %.2fms (%.2f MVPS) %ux%ux%u Nv=%u Ni=%u ix=%s memfree=%zumb/%zumb",
                 frames / s, (float(frames)* field_size.x* field_size.y* field_size.z) / (1000000.f * s),
                 cuda_ms / frames, (float(frames)* field_size.x* field_size.y* field_size.z) / (1000.f * cuda_ms),
                 field_size.x, field_size.y, field_size.z,
                 vertex_count,
                 index_count,
                 indexed ? "y" : "n",
                 (free + 1024 * 1024 - 1) / (1024 * 1024),
                 (total + 1024 * 1024 - 1) / (1024 * 1024));
        timer = now;
        frames = 0;
        cuda_ms = 0.f;
      }
    }


  }
  glfwDestroyWindow(win);
  glfwTerminate();

  return EXIT_SUCCESS;
}
